#include "hip/hip_runtime.h"
//********************************************************//
// CUDA SIFT extractor by Marten Bjorkman aka Celebrandil //
//********************************************************//  

#include <cudautils.h>
#include "cudaSiftD.h"
#include "cudaSift.h"

///////////////////////////////////////////////////////////////////////////////
// Kernel configuration
///////////////////////////////////////////////////////////////////////////////

__constant__ float d_Threshold[2];
__constant__ float d_Scales[8], d_Factor;
__constant__ float d_EdgeLimit;
__constant__ int d_MaxNumPoints;

__device__ unsigned int d_PointCounter[1];

texture<float, 2, hipReadModeElementType> tex;

///////////////////////////////////////////////////////////////////////////////
// Lowpass filter an subsample image
///////////////////////////////////////////////////////////////////////////////
__global__ void ScaleDown(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
  __shared__ float inrow[SCALEDOWN_W+4]; 
  __shared__ float brow[5*(SCALEDOWN_W/2)];
  __shared__ int yRead[SCALEDOWN_H+4], yWrite[SCALEDOWN_H+4];
  #define dx2 (SCALEDOWN_W/2)
  const int tx = threadIdx.x;
  const int tx0 = tx + 0*dx2;
  const int tx1 = tx + 1*dx2;
  const int tx2 = tx + 2*dx2;
  const int tx3 = tx + 3*dx2;
  const int tx4 = tx + 4*dx2;
  const int xStart = blockIdx.x*SCALEDOWN_W;
  const int yStart = blockIdx.y*SCALEDOWN_H;
  const int xWrite = xStart/2 + tx;
  const float *k = d_Kernel;
  if (tx<SCALEDOWN_H+4) {
    int y = yStart + tx - 1;
    y = (y<0 ? 0 : y);
    y = (y>=height ? height-1 : y);
    yRead[tx] = y*pitch;
    yWrite[tx] = (yStart + tx - 4)/2 * newpitch;
  }
  __syncthreads();
  int xRead = xStart + tx - WARP_SIZE;
  xRead = (xRead<0 ? 0 : xRead);
  xRead = (xRead>=width ? width-1 : xRead);
  for (int dy=0;dy<SCALEDOWN_H+4;dy+=5) {
    if (tx>=WARP_SIZE-2) 
      inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+0] + xRead];
    __syncthreads();
    if (tx<dx2) 
      brow[tx0] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
    __syncthreads();
    if (tx<dx2 && dy>=4 && !(dy&1)) 
      d_Result[yWrite[dy+0] + xWrite] = k[2]*brow[tx2] + k[0]*(brow[tx0]+brow[tx4]) + k[1]*(brow[tx1]+brow[tx3]);
    if (dy<(SCALEDOWN_H+3)) {
      if (tx>=WARP_SIZE-2) 
	inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+1] + xRead];
      __syncthreads();
      if (tx<dx2)
	brow[tx1] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
      __syncthreads();
      if (tx<dx2 && dy>=3 && (dy&1)) 
	d_Result[yWrite[dy+1] + xWrite] = k[2]*brow[tx3] + k[0]*(brow[tx1]+brow[tx0]) + k[1]*(brow[tx2]+brow[tx4]); 
    }
    if (dy<(SCALEDOWN_H+2)) {
      if (tx>=WARP_SIZE-2) 
	inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+2] + xRead];
      __syncthreads();
      if (tx<dx2)
	brow[tx2] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
      __syncthreads();
      if (tx<dx2 && dy>=2 && !(dy&1)) 
	d_Result[yWrite[dy+2] + xWrite] = k[2]*brow[tx4] + k[0]*(brow[tx2]+brow[tx1]) + k[1]*(brow[tx3]+brow[tx0]); 
    }
    if (dy<(SCALEDOWN_H+1)) {
      if (tx>=WARP_SIZE-2) 
	inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+3] + xRead];
      __syncthreads();
      if (tx<dx2)
	brow[tx3] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
      __syncthreads();
      if (tx<dx2 && dy>=1 && (dy&1)) 
	d_Result[yWrite[dy+3] + xWrite] = k[2]*brow[tx0] + k[0]*(brow[tx3]+brow[tx2]) + k[1]*(brow[tx4]+brow[tx1]); 
    }
    if (dy<SCALEDOWN_H) {
      if (tx>=WARP_SIZE-2) 
	inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+4] + xRead];
      __syncthreads();
      if (tx<dx2)
	brow[tx4] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
      __syncthreads();
      if (tx<dx2 && !(dy&1)) 
	d_Result[yWrite[dy+4] + xWrite] = k[2]*brow[tx1] + k[0]*(brow[tx4]+brow[tx3]) + k[1]*(brow[tx0]+brow[tx2]); 
    }
    __syncthreads();
  }
}

///////////////////////////////////////////////////////////////////////////////
// Subtract two images
///////////////////////////////////////////////////////////////////////////////
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
  const int x = blockIdx.x*SUBTRACT_W + threadIdx.x;
  const int y = blockIdx.y*SUBTRACT_H + threadIdx.y;
  int p = y*pitch + x;
  if (x<width && y<height)
    d_Result[p] = d_Data1[p] - d_Data2[p];
  __syncthreads();
}

///////////////////////////////////////////////////////////////////////////////
// Extract Sift descriptors
///////////////////////////////////////////////////////////////////////////////
__global__ void ExtractSiftDescriptors(float *g_Data, float *d_sift, float *d_desc, int maxPts)
{
  __shared__ float buffer[NUMDESCBUFS*128];
  __shared__ float gauss[16];
  __shared__ float gradients[256];
  __shared__ float angles[256];
  const int tx = threadIdx.x; // 0 -> 16
  const int bx = blockIdx.x;  // 0 -> numPts
  gauss[tx] = exp(-(tx-7.5f)*(tx-7.5f)/128.0f);
  __syncthreads();
  float theta = 2.0f*3.1415f/360.0f*d_sift[5*maxPts+bx];
  float sina = sinf(theta);           // cosa -sina
  float cosa = cosf(theta);           // sina  cosa
  float scale = 12.0f/16.0f*d_sift[2*maxPts+bx];
  float ssina = scale*sina;
  float scosa = scale*cosa;
  // Compute angles and gradients
  float xpos = d_sift[0*maxPts+bx] + (tx-7.5f)*scosa + 7.5f*ssina;
  float ypos = d_sift[1*maxPts+bx] + (tx-7.5f)*ssina - 7.5f*scosa;
  for (int i=0;i<128*NUMDESCBUFS/16;i++)
    buffer[16*i+tx] = 0.0f;
  for (int y=0;y<16;y++) {
    float dx = tex2D(tex, xpos+cosa, ypos+sina) - tex2D(tex, xpos-cosa, ypos-sina);
    float dy = tex2D(tex, xpos-sina, ypos+cosa) - tex2D(tex, xpos+sina, ypos-cosa);
    gradients[16*y+tx] = gauss[y]*gauss[tx] * sqrtf(dx*dx + dy*dy);
    angles[16*y+tx] = 4.0f/3.1415f*atan2f(dy, dx) + 4.0f;
    xpos -= ssina;
    ypos += scosa;
  }
  __syncthreads();
  if (tx<NUMDESCBUFS) {
    for (int txi=tx;txi<16;txi+=NUMDESCBUFS) {
      int hori = (txi + 2)/4 - 1;
      float horf = (txi - 1.5f)/4.0f - hori;
      float ihorf = 1.0f - horf;
      int veri = -1;
      float verf = 1.0f - 1.5f/4.0f;
      for (int y=0;y<16;y++) {
	int i = 16*y + txi;
	float grad = gradients[i];
	float angf = angles[i];
	int angi = angf;
	int angp = (angi<7 ? angi+1 : 0);
	angf -= angi;
	float iangf = 1.0f - angf;
	float iverf = 1.0f - verf;
	int hist = 8*(4*veri + hori);
	//printf("%d\n", hist);
	int p1 = tx + NUMDESCBUFS*(angi+hist);
	int p2 = tx + NUMDESCBUFS*(angp+hist);
	if (txi>=2) { 
	  float grad1 = ihorf*grad;
	  if (y>=2) {
	    float grad2 = iverf*grad1;
	    buffer[p1+0] += iangf*grad2;
	    buffer[p2+0] +=  angf*grad2;
	  }
	  if (y<=14) {
	    float grad2 = verf*grad1;
	    buffer[p1+32*NUMDESCBUFS] += iangf*grad2; 
	    buffer[p2+32*NUMDESCBUFS] +=  angf*grad2;
	  }
	}
	if (txi<=14) { 
	  float grad1 = horf*grad;
	  if (y>=2) {
	    float grad2 = iverf*grad1;
	    buffer[p1+8*NUMDESCBUFS] += iangf*grad2;
	    buffer[p2+8*NUMDESCBUFS] +=  angf*grad2;
	  }
	  if (y<=14) {
	    float grad2 = verf*grad1;
	    buffer[p1+40*NUMDESCBUFS] += iangf*grad2;
	    buffer[p2+40*NUMDESCBUFS] +=  angf*grad2;
	  }
	}
	verf += 0.25f;
	if (verf>1.0f) {
	  verf -= 1.0f;
	  veri ++;
	}
      }
    }
  }
  __syncthreads();
  const int t2 = (tx&14)*8;
  const int tx2 = (tx&1);
  for (int i=0;i<16;i++) 
    buffer[NUMDESCBUFS*(i+t2)+tx2] += buffer[NUMDESCBUFS*(i+t2)+tx2+2];
  __syncthreads();

  const int t1 = tx*8;                 
  const int bptr = NUMDESCBUFS*tx + 2;   
  buffer[bptr] = 0.0f;
  for (int i=0;i<8;i++) {
    int p = NUMDESCBUFS*(i+t1);  
    buffer[p] += buffer[p+1];
    buffer[bptr] += buffer[p]*buffer[p];
  }
  __syncthreads();

  if (tx<8) 
    buffer[bptr] += buffer[bptr+8*NUMDESCBUFS];
  __syncthreads();
  if (tx<4) 
    buffer[bptr] += buffer[bptr+4*NUMDESCBUFS];
  __syncthreads();
  if (tx<2) 
    buffer[bptr] += buffer[bptr+2*NUMDESCBUFS];
  __syncthreads();
  float isum = 1.0f / sqrt(buffer[2] + buffer[NUMDESCBUFS+2]);

  buffer[bptr] = 0.0f;
  for (int i=0;i<8;i++) {
    int p = NUMDESCBUFS*(i+t1);
    buffer[p] = isum*buffer[p];
    if (buffer[p]>0.2f)
      buffer[p] = 0.2f;
    buffer[bptr] += buffer[p]*buffer[p];
  }
  __syncthreads();

  if (tx<8) 
    buffer[bptr] += buffer[bptr+8*NUMDESCBUFS];
  __syncthreads();
  if (tx<4) 
    buffer[bptr] += buffer[bptr+4*NUMDESCBUFS];
  __syncthreads();
  if (tx<2) 
    buffer[bptr] += buffer[bptr+2*NUMDESCBUFS];
  __syncthreads();
  isum = 1.0f / sqrt(buffer[2] + buffer[NUMDESCBUFS+2]);

  for (int i=0;i<8;i++) {
    int p = NUMDESCBUFS*(i+t1);
    d_desc[128*bx+(i+t1)] = isum*buffer[p];
  }
}
 
///============= New functions

#if 1

__global__ void FindPoints(float *d_Data1, float *d_Data2, float *d_Data3, float *d_Sift, int width, int pitch, int height)
{
  #define MEMWID (MINMAX_W + 2)
  __shared__ float data1[3*MEMWID], data2[3*MEMWID], data3[3*MEMWID];
  __shared__ float ymin1[MEMWID],   ymin2[MEMWID],   ymin3[MEMWID];
  __shared__ float ymax1[MEMWID],   ymax2[MEMWID],   ymax3[MEMWID];

  const int tx = threadIdx.x;
  const int minx = blockIdx.x*MINMAX_W;
  const int maxx = min(minx + MINMAX_W, width);
  const int xpos = minx + tx;

  int ptr0 = tx;
  int ptr1 = tx;
  int yq = 0;
  for (int y=0;y<MINMAX_H+2;y++) {

    int ypos = MINMAX_H*blockIdx.y + y - 1;
    int yptr = min(max(ypos, 0), height - 1)*pitch;
    int xposr = xpos - 1;
    int ptr2 = yq*MEMWID + tx;

    if (xposr<0) {
      data1[ptr2] = 0;
      data2[ptr2] = 0;
      data3[ptr2] = 0;
    } else if (xposr>=width) {
      data1[ptr2] = 0;
      data2[ptr2] = 0;
      data3[ptr2] = 0;
    } else {
      data1[ptr2] = d_Data1[yptr + xposr];
      data2[ptr2] = d_Data2[yptr + xposr];
      data3[ptr2] = d_Data3[yptr + xposr];
    }
    //__syncthreads();
  
    if (y>1) {
      float min1 = fminf(fminf(data1[ptr0], data1[ptr1]), data1[ptr2]);
      float min2 = fminf(fminf(data2[ptr0], data2[ptr1]), data2[ptr2]);
      float min3 = fminf(fminf(data3[ptr0], data3[ptr1]), data3[ptr2]);
      float max1 = fmaxf(fmaxf(data1[ptr0], data1[ptr1]), data1[ptr2]);
      float max2 = fmaxf(fmaxf(data2[ptr0], data2[ptr1]), data2[ptr2]);
      float max3 = fmaxf(fmaxf(data3[ptr0], data3[ptr1]), data3[ptr2]);
      ymin1[tx] = min1;
      ymin2[tx] = fminf(fminf(min1, min2), min3);
      ymin3[tx] = min3;
      ymax1[tx] = max1;
      ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
      ymax3[tx] = max3;
    }
    //__syncthreads();

    if (y>1) {
      if (tx<MINMAX_W && xpos<maxx) {
	float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx+2]), ymin1[tx+1]), ymin3[tx+1]), data2[ptr0+1]), data2[ptr2+1]);
	minv = fminf(minv, d_Threshold[1]);
	float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx+2]), ymax1[tx+1]), ymax3[tx+1]), data2[ptr0+1]), data2[ptr2+1]);
	maxv = fmaxf(maxv, d_Threshold[0]);
	float val = data2[ptr1+1];
	if (val<minv || val>maxv) {
	  float dxx = 2.0f*val - data2[ptr1+0] - data2[ptr1+2];
	  float dyy = 2.0f*val - data2[ptr0+1] - data2[ptr2+1];
	  float dxy = 0.25f*(data2[ptr2+2] + data2[ptr0+0] - data2[ptr0+2] - data2[ptr2+0]);
	  float tra = dxx + dyy;
	  float det = dxx*dyy - dxy*dxy;
	  if (tra*tra<d_EdgeLimit*det) {
	    float edge = __fdividef(tra*tra, det);
	    float dx = 0.5f*(data2[ptr1+2] - data2[ptr1+0]);
	    float dy = 0.5f*(data2[ptr2+1] - data2[ptr0+1]); 
	    float ds = 0.5f*(data1[ptr1+1] - data3[ptr1+1]); 
	    float dss = 2.0f*val - data3[ptr1+1] - data1[ptr1+1];
	    float dxs = 0.25f*(data3[ptr1+2] + data1[ptr1+0] - data1[ptr1+2] - data3[ptr1+0]);
	    float dys = 0.25f*(data3[ptr2+1] + data1[ptr0+1] - data3[ptr0+1] - data1[ptr2+1]);
	    float idxx = dyy*dss - dys*dys;
	    float idxy = dys*dxs - dxy*dss;  
	    float idxs = dxy*dys - dyy*dxs;
	    float idyy = dxx*dss - dxs*dxs;
	    float idys = dxy*dxs - dxx*dys;
	    float idss = dxx*dyy - dxy*dxy;
	    float idet = __fdividef(1.0f, idxx*dxx + idxy*dxy + idxs*dxs);
	    float pdx = idet*(idxx*dx + idxy*dy + idxs*ds);
	    float pdy = idet*(idxy*dx + idyy*dy + idys*ds);
	    float pds = idet*(idxs*dx + idys*dy + idss*ds);
	    if (pdx<-0.5f || pdx>0.5f || pdy<-0.5f || pdy>0.5f || pds<-0.5f || pds>0.5f) {
	      pdx = __fdividef(dx, dxx);
	      pdy = __fdividef(dy, dyy);
	      pds = __fdividef(ds, dss);
	    }
	    float dval = 0.5f*(dx*pdx + dy*pdy + ds*pds);
	    int maxPts = d_MaxNumPoints;
	    unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
	    idx = (idx>=maxPts ? maxPts-1 : idx);
	    d_Sift[idx + 0*maxPts] = xpos + pdx;
	    d_Sift[idx + 1*maxPts] = ypos - 1 + pdy;
	    d_Sift[idx + 2*maxPts] = d_Scales[0] * exp2f(pds*d_Factor);
	    d_Sift[idx + 3*maxPts] = val + dval;
	    d_Sift[idx + 4*maxPts] = edge;
	    //printf("idx: %d %.1f %.1f %.2f\n", idx, d_Sift[idx + 0*maxPts], d_Sift[idx + 1*maxPts], edge);
	  }
	}
      }
    }
    __syncthreads();

    ptr0 = ptr1;
    ptr1 = ptr2;
    yq = (yq<2 ? yq+1 : 0);
  }
}

#else

__global__ void FindPoints(float *d_Data1, float *d_Data2, float *d_Data3, float *d_Sift, int width, int pitch, int height)
{
  #define MEMWID (MINMAX_W + 2)
  __shared__ float data1[3*MEMWID], data2[3*MEMWID], data3[3*MEMWID];
  __shared__ float ymin1[MEMWID],   ymin2[MEMWID],   ymin3[MEMWID];
  __shared__ float ymax1[MEMWID],   ymax2[MEMWID],   ymax3[MEMWID];

  const int tx = threadIdx.x;
  const int minx = blockIdx.x*MINMAX_W;
  const int maxx = min(minx + MINMAX_W, width);

  int ptr0 = 0;
  int ptr1 = 0;
  int yq = 0;
  for (int y=0;y<MINMAX_H+2;y++) {

    int ypos = MINMAX_H*blockIdx.y + y - 1;
    int yptr = min(max(ypos, 0), height - 1)*pitch;
    int ptr2 = yq*MEMWID;

    for (int idx=tx;idx<MEMWID;idx+=MINMAX_S) {
      int xpos = minx - 1 + idx;
      int p0 = ptr0 + idx;
      int p1 = ptr1 + idx;
      int p2 = ptr2 + idx;
      if (xpos<0) {
	data1[p2] = 0;
	data2[p2] = 0;
	data3[p2] = 0;
      } else if (xpos>=width) {
	data1[p2] = 0;
	data2[p2] = 0;
	data3[p2] = 0;
      } else {
	data1[p2] = d_Data1[yptr + xpos];
	data2[p2] = d_Data2[yptr + xpos];
	data3[p2] = d_Data3[yptr + xpos];
      }
      if (y>1) {
	float min1 = fminf(fminf(data1[p0], data1[p1]), data1[p2]);
	float min2 = fminf(fminf(data2[p0], data2[p1]), data2[p2]);
	float min3 = fminf(fminf(data3[p0], data3[p1]), data3[p2]);
	float max1 = fmaxf(fmaxf(data1[p0], data1[p1]), data1[p2]);
	float max2 = fmaxf(fmaxf(data2[p0], data2[p1]), data2[p2]);
	float max3 = fmaxf(fmaxf(data3[p0], data3[p1]), data3[p2]);
	ymin1[idx] = min1;
	ymin2[idx] = fminf(fminf(min1, min2), min3);
	ymin3[idx] = min3;
	ymax1[idx] = max1;
	ymax2[idx] = fmaxf(fmaxf(max1, max2), max3);
	ymax3[idx] = max3;
      }
    }
    __syncthreads();

    if (y>1) {
      for (int idx=tx;idx<MINMAX_W && (minx+idx)<maxx;idx+=MINMAX_S) {
	const int xpos = minx + idx;
	int p0 = ptr0 + idx;
	int p1 = ptr1 + idx;
	int p2 = ptr2 + idx;
	float minv = fminf(fminf(fminf(fminf(fminf(ymin2[idx], ymin2[idx+2]), ymin1[idx+1]), ymin3[idx+1]), data2[p0+1]), data2[p2+1]);
	minv = fminf(minv, d_Threshold[1]);
	float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[idx], ymax2[idx+2]), ymax1[idx+1]), ymax3[idx+1]), data2[p0+1]), data2[p2+1]);
	maxv = fmaxf(maxv, d_Threshold[0]);
	float val = data2[p1+1];
	if (val<minv || val>maxv) {
	  float dxx = 2.0f*val - data2[p1+0] - data2[p1+2];
	  float dyy = 2.0f*val - data2[p0+1] - data2[p2+1];
	  float dxy = 0.25f*(data2[p2+2] + data2[p0+0] - data2[p0+2] - data2[p2+0]);
	  float tra = dxx + dyy;
	  float det = dxx*dyy - dxy*dxy;
	  if (tra*tra<d_EdgeLimit*det) {
	    float edge = __fdividef(tra*tra, det);
	    float dx = 0.5f*(data2[p1+2] - data2[p1+0]);
	    float dy = 0.5f*(data2[p2+1] - data2[p0+1]); 
	    float ds = 0.5f*(data1[p1+1] - data3[p1+1]); 
	    float dss = 2.0f*val - data3[p1+1] - data1[p1+1];
	    float dxs = 0.25f*(data3[p1+2] + data1[p1+0] - data1[p1+2] - data3[p1+0]);
	    float dys = 0.25f*(data3[p2+1] + data1[p0+1] - data3[p0+1] - data1[p2+1]);
	    float idxx = dyy*dss - dys*dys;
	    float idxy = dys*dxs - dxy*dss;  
	    float idxs = dxy*dys - dyy*dxs;
	    float idyy = dxx*dss - dxs*dxs;
	    float idys = dxy*dxs - dxx*dys;
	    float idss = dxx*dyy - dxy*dxy;
	    float idet = __fdividef(1.0f, idxx*dxx + idxy*dxy + idxs*dxs);
	    float pdx = idet*(idxx*dx + idxy*dy + idxs*ds);
	    float pdy = idet*(idxy*dx + idyy*dy + idys*ds);
	    float pds = idet*(idxs*dx + idys*dy + idss*ds);
	    if (pdx<-0.5f || pdx>0.5f || pdy<-0.5f || pdy>0.5f || pds<-0.5f || pds>0.5f) {
	      pdx = __fdividef(dx, dxx);
	      pdy = __fdividef(dy, dyy);
	      pds = __fdividef(ds, dss);
	    }
	    float dval = 0.5f*(dx*pdx + dy*pdy + ds*pds);
	    int maxPts = d_MaxNumPoints;
	    unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
	    idx = (idx>=maxPts ? maxPts-1 : idx);
	    d_Sift[i + 0*maxPts] = xpos + pdx;
	    d_Sift[i + 1*maxPts] = ypos - 1 + pdy;
	    d_Sift[i + 2*maxPts] = d_Scales[0] * exp2f(pds*d_Factor);
	    d_Sift[i + 3*maxPts] = val + dval;
	    d_Sift[i + 4*maxPts] = edge;
	    //printf("i: %d %.1f %.1f %.2f\n", i, d_Sift[i + 0*maxPts], d_Sift[i + 1*maxPts], edge);
	  }
	}
      }
    }
    __syncthreads();

    ptr0 = ptr1;
    ptr1 = ptr2;
    yq = (yq<2 ? yq+1 : 0);
  }
}

#endif

__global__ void ComputeOrientations(float *g_Data, float *d_Sift, int maxPts, int w, int h)
{
  __shared__ float data[16*15];
  __shared__ float hist[32*13];
  __shared__ float gauss[16];
  const int tx = threadIdx.x;
  const int bx = blockIdx.x;
  for (int i=0;i<13;i++)
    hist[i*32+tx] = 0.0f;
  __syncthreads();
  float i2sigma2 = -1.0f/(2.0f*3.0f*3.0f);
  if (tx<15) 
    gauss[tx] = exp(i2sigma2*(tx-7)*(tx-7));
  int xp = (int)(d_Sift[bx + 0*maxPts] - 6.5f);
  int yp = (int)(d_Sift[bx + 1*maxPts] - 6.5f);
  int px = xp & 15;
  int x = tx - px;

  for (int y=0;y<15;y++) {
    int memPos = 16*y + x;
    int xi = xp + x;
    int yi = yp + y;
    if (xi<0) xi = 0;
    if (xi>=w) xi = w-1;
    if (yi<0) yi = 0;
    if (yi>=h) yi = h-1;
    if (x>=0 && x<15) 
      data[memPos] = g_Data[yi*w+xi];
  }
  __syncthreads();
  for (int y=1;y<14;y++) {
    int memPos = 16*y + x;
    if (x>=1 && x<14) {
      float dy = data[memPos+16] - data[memPos-16];
      float dx = data[memPos+1]  - data[memPos-1];
      int bin = 16.0f*atan2f(dy, dx)/3.1416f + 16.5f;
      if (bin==32)
	bin = 0;
      float grad = sqrtf(dx*dx + dy*dy);
      hist[32*(x-1)+bin] += grad*gauss[x]*gauss[y];
    }
  }
  __syncthreads();
  for (int y=0;y<5;y++)
    hist[y*32+tx] += hist[(y+8)*32+tx];
  __syncthreads();
  for (int y=0;y<4;y++)
    hist[y*32+tx] += hist[(y+4)*32+tx];
  __syncthreads();
  for (int y=0;y<2;y++)
    hist[y*32+tx] += hist[(y+2)*32+tx];
  __syncthreads();
  hist[tx] += hist[32+tx];
  __syncthreads();
  if (tx==0) 
    hist[32] = 6*hist[0] + 4*(hist[1]+hist[31]) + (hist[2]+hist[30]);
  if (tx==1)
    hist[33] = 6*hist[1] + 4*(hist[2]+hist[0]) + (hist[3]+hist[31]);
  if (tx>=2 && tx<=29)
    hist[tx+32] = 6*hist[tx] + 4*(hist[tx+1]+hist[tx-1]) + 
      (hist[tx+2]+hist[tx-2]);
  if (tx==30)
    hist[62] = 6*hist[30] + 4*(hist[31]+hist[29]) + (hist[0]+hist[28]);
  if (tx==31)
    hist[63] = 6*hist[31] + 4*(hist[0]+hist[30]) + (hist[1]+hist[29]);
  __syncthreads();
  float v = hist[32+tx];
  hist[tx] = (v>hist[32+((tx+1)&31)] && v>=hist[32+((tx+31)&31)] ? v : 0.0f);
  __syncthreads();
  if (tx==0) {
    float maxval1 = 0.0;
    float maxval2 = 0.0;
    int i1 = -1;
    int i2 = -1;
    for (int i=0;i<32;i++) {
      float v = hist[i];
      if (v>maxval1) {
	maxval2 = maxval1;
	maxval1 = v;
	i2 = i1;
	i1 = i;
      } else if (v>maxval2) {
	maxval2 = v;
	i2 = i;
      }
    }
    float val1 = hist[32+((i1+1)&31)];
    float val2 = hist[32+((i1+31)&31)];
    float peak = i1 + 0.5f*(val1-val2) / (2.0f*maxval1-val1-val2);
    d_Sift[bx + 5*maxPts] = 11.25f*(peak<0.0f ? peak+32.0f : peak);
    if (maxval2<0.8f*maxval1) 
      i2 = -1;
    if (i2>=0) {
      float val1 = hist[32+((i2+1)&31)];
      float val2 = hist[32+((i2+31)&31)];
      float peak = i2 + 0.5f*(val1-val2) / (2.0f*maxval2-val1-val2);
      d_Sift[bx + 6*maxPts] = 11.25f*(peak<0.0f ? peak+32.0f : peak);;
    } else 
      d_Sift[bx + 6*maxPts] = i2;
  }
} 

///////////////////////////////////////////////////////////////////////////////
// Subtract two images (multi-scale version)
///////////////////////////////////////////////////////////////////////////////

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
  const int x = blockIdx.x*SUBTRACTM_W + threadIdx.x;
  const int y = blockIdx.y*SUBTRACTM_H + threadIdx.y;
  int sz = height*pitch;
  int p = threadIdx.z*sz + y*pitch + x;
  if (x<width && y<height)
    d_Result[p] = d_Data[p] - d_Data[p + sz];
  __syncthreads();
}

__global__ void FindPointsMulti(float *d_Data0, float *d_Sift, int width, int pitch, int height, int nScales)
{
  #define MEMWID (MINMAX_W + 2)
  __shared__ float data1[3*MEMWID], data2[3*MEMWID], data3[3*MEMWID];
  __shared__ float ymin1[MEMWID],   ymin2[MEMWID],   ymin3[MEMWID];
  __shared__ float ymax1[MEMWID],   ymax2[MEMWID],   ymax3[MEMWID];

  const int tx = threadIdx.x;
  const int block = blockIdx.x/nScales; 
  const int scale = blockIdx.x - nScales*block;
  const int minx = block*MINMAX_W;
  const int maxx = min(minx + MINMAX_W, width);
  const int xpos = minx + tx;
  const int size = pitch*height;
  const float *d_Data1 = d_Data0 + size*scale;
  const float *d_Data2 = d_Data1 + size;
  const float *d_Data3 = d_Data2 + size;
  //if (block==0 && blockIdx.y==0 && tx==0)
  //  printf("%08x %08x %08x %d %d\n", d_Data1, d_Data2, d_Data3, pitch, height);

  int ptr0 = tx;
  int ptr1 = tx;
  int yq = 0;
  for (int y=0;y<MINMAX_H+2;y++) {

    int xposr = xpos - 1;
    int ypos = MINMAX_H*blockIdx.y + y - 1;
    int yptr = min(max(ypos, 0), height - 1)*pitch;

    int ptr2 = yq*MEMWID + tx;
    if (xposr<0) {
      data1[ptr2] = 0;
      data2[ptr2] = 0;
      data3[ptr2] = 0;
    } else if (xposr>=width) {
      data1[ptr2] = 0;
      data2[ptr2] = 0;
      data3[ptr2] = 0;
    } else {
      data1[ptr2] = d_Data1[yptr + xposr];
      data2[ptr2] = d_Data2[yptr + xposr];
      data3[ptr2] = d_Data3[yptr + xposr];
    }
    //__syncthreads();
    if (y>1) {
      float min1 = fminf(fminf(data1[ptr0], data1[ptr1]), data1[ptr2]);
      float min2 = fminf(fminf(data2[ptr0], data2[ptr1]), data2[ptr2]);
      float min3 = fminf(fminf(data3[ptr0], data3[ptr1]), data3[ptr2]);
      float max1 = fmaxf(fmaxf(data1[ptr0], data1[ptr1]), data1[ptr2]);
      float max2 = fmaxf(fmaxf(data2[ptr0], data2[ptr1]), data2[ptr2]);
      float max3 = fmaxf(fmaxf(data3[ptr0], data3[ptr1]), data3[ptr2]);
      ymin1[tx] = min1;
      ymin2[tx] = fminf(fminf(min1, min2), min3);
      ymin3[tx] = min3;
      ymax1[tx] = max1;
      ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
      ymax3[tx] = max3;
    }
    //__syncthreads();
    if (y>1) {
      if (tx<MINMAX_W && xpos<maxx) {
	float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx+2]), ymin1[tx+1]), ymin3[tx+1]), data2[ptr0+1]), data2[ptr2+1]);
	minv = fminf(minv, d_Threshold[1]);
	float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx+2]), ymax1[tx+1]), ymax3[tx+1]), data2[ptr0+1]), data2[ptr2+1]);
	maxv = fmaxf(maxv, d_Threshold[0]);
	float val = data2[ptr1+1];
	if (val<minv || val>maxv) {
	  float dxx = 2.0f*val - data2[ptr1+0] - data2[ptr1+2];
	  float dyy = 2.0f*val - data2[ptr0+1] - data2[ptr2+1];
	  float dxy = 0.25f*(data2[ptr2+2] + data2[ptr0+0] - data2[ptr0+2] - data2[ptr2+0]);
	  float tra = dxx + dyy;
	  float det = dxx*dyy - dxy*dxy;
	  if (tra*tra<d_EdgeLimit*det) {
	    float edge = __fdividef(tra*tra, det);
	    float dx = 0.5f*(data2[ptr1+2] - data2[ptr1+0]);
	    float dy = 0.5f*(data2[ptr2+1] - data2[ptr0+1]); 
	    float ds = 0.5f*(data1[ptr1+1] - data3[ptr1+1]); 
	    float dss = 2.0f*val - data3[ptr1+1] - data1[ptr1+1];
	    float dxs = 0.25f*(data3[ptr1+2] + data1[ptr1+0] - data1[ptr1+2] - data3[ptr1+0]);
	    float dys = 0.25f*(data3[ptr2+1] + data1[ptr0+1] - data3[ptr0+1] - data1[ptr2+1]);
	    float idxx = dyy*dss - dys*dys;
	    float idxy = dys*dxs - dxy*dss;  
	    float idxs = dxy*dys - dyy*dxs;
	    float idyy = dxx*dss - dxs*dxs;
	    float idys = dxy*dxs - dxx*dys;
	    float idss = dxx*dyy - dxy*dxy;
	    float idet = __fdividef(1.0f, idxx*dxx + idxy*dxy + idxs*dxs);
	    float pdx = idet*(idxx*dx + idxy*dy + idxs*ds);
	    float pdy = idet*(idxy*dx + idyy*dy + idys*ds);
	    float pds = idet*(idxs*dx + idys*dy + idss*ds);
	    if (pdx<-0.5f || pdx>0.5f || pdy<-0.5f || pdy>0.5f || pds<-0.5f || pds>0.5f) {
	      pdx = __fdividef(dx, dxx);
	      pdy = __fdividef(dy, dyy);
	      pds = __fdividef(ds, dss);
	    }
	    float dval = 0.5f*(dx*pdx + dy*pdy + ds*pds);
	    int maxPts = d_MaxNumPoints;
	    unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
	    idx = (idx>=maxPts ? maxPts-1 : idx);
	    d_Sift[idx + 0*maxPts] = xpos + pdx;
	    d_Sift[idx + 1*maxPts] = ypos - 1 + pdy;
	    d_Sift[idx + 2*maxPts] = d_Scales[scale] * exp2f(pds*d_Factor);
	    d_Sift[idx + 3*maxPts] = val + dval;
	    d_Sift[idx + 4*maxPts] = edge;
	  }
	}
      }
    }
    __syncthreads();
    ptr0 = ptr1;
    ptr1 = ptr2;
    yq = (yq<2 ? yq+1 : 0);
  }
}

#define RADIUS 4

__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
  __shared__ float data[CONVROW_W + 2*RADIUS];
  const int tx = threadIdx.x;
  const int block = blockIdx.x/(NUM_SCALES+3); 
  const int scale = blockIdx.x - (NUM_SCALES+3)*block;
  const int xout = block*CONVROW_W + tx;
  const int loadPos = xout - RADIUS; 
  const int yptr = blockIdx.y*pitch;
  const int writePos = yptr + height*pitch*scale + xout;
  float *kernel = d_Kernel + scale*16;

  if (loadPos<0) 
    data[tx] = d_Data[yptr];
  else if (loadPos>=width) 
    data[tx] = d_Data[yptr + width-1];
  else
    data[tx] = d_Data[yptr + loadPos];
  __syncthreads();
  if (xout<width && tx<CONVROW_W) 
    d_Result[writePos] = 
      (data[tx+0] + data[tx+8])*kernel[0] + 
      (data[tx+1] + data[tx+7])*kernel[1] + 
      (data[tx+2] + data[tx+6])*kernel[2] + 
      (data[tx+3] + data[tx+5])*kernel[3] + 
      data[tx+4]*kernel[4]; 
  __syncthreads();
}

__global__ void LowPassColMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
  __shared__ float data[CONVCOL_W*(CONVCOL_H + 2*RADIUS)];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int block = blockIdx.x/(NUM_SCALES+3); 
  const int scale = blockIdx.x - (NUM_SCALES+3)*block;
  const int miny = blockIdx.y*CONVCOL_H;
  const int maxy = min(miny + CONVCOL_H, height) - 1;
  const int totStart = miny - RADIUS;
  const int totEnd = maxy + RADIUS;
  const int colStart = block*CONVCOL_W + tx;
  const int colEnd = colStart + (height-1)*pitch;
  const int sStep = CONVCOL_W*CONVCOL_S;
  const int gStep = pitch*CONVCOL_S;
  float *kernel = d_Kernel + scale*16;
  const int size = pitch*height*scale;
  d_Result += size;
  d_Data += size;
 
  if (colStart<width) {
    float *sdata = data + ty*CONVCOL_W + tx;
    int gPos = colStart + (totStart + ty)*pitch;
    for (int y = totStart+ty;y<=totEnd;y+=blockDim.y){
      if (y<0) 
	sdata[0] = d_Data[colStart];
      else if (y>=height) 
	sdata[0] = d_Data[colEnd];
      else 
	sdata[0] = d_Data[gPos];  
      sdata += sStep;
      gPos += gStep;
    }
  }
  __syncthreads();
  if (colStart<width) {
    float *sdata = data + ty*CONVCOL_W + tx;
    int gPos = colStart + (miny + ty)*pitch;
    for (int y=miny+ty;y<=maxy;y+=blockDim.y) {
      d_Result[gPos] = 
	(sdata[0*CONVCOL_W] + sdata[8*CONVCOL_W])*kernel[0] + 
	(sdata[1*CONVCOL_W] + sdata[7*CONVCOL_W])*kernel[1] + 
	(sdata[2*CONVCOL_W] + sdata[6*CONVCOL_W])*kernel[2] + 
	(sdata[3*CONVCOL_W] + sdata[5*CONVCOL_W])*kernel[3] + 
	sdata[4*CONVCOL_W]*kernel[4]; 
      sdata += sStep;
      gPos += gStep;
    }
  }
}
